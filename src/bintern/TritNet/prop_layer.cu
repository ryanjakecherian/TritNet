#include "TritNet.hpp"

#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <ctime>

using namespace std;

template<typename T>
void TritNet<T>::propagate_layer(int i, T *&d_A, T *&d_W, T *&d_O){ //pytorch and tensorflow convention is Y = X W because it is more efficient to store each individual image as a row, than as a column.
    
    if (hipSuccess!= hipMemcpy(d_A, A_list[i], n*A_bytesizes[i]+1, hipMemcpyHostToDevice) ) {throw std::runtime_error("CUDA memcpy failed");};
    if (hipSuccess!= hipMemcpy(d_W, W_list[i], W_bytesizes[i], hipMemcpyHostToDevice) ) {throw std::runtime_error("CUDA memcpy failed");};
    
    if (hipSuccess!= hipMemset(d_O, 0, n*A_bytesizes[i+1]+1) ) {throw std::runtime_error("CUDA memset failed");};
    
    dim3 blockDim = WORD_SIZE;                 
    dim3 gridDim(layers[i+1], n);       //i.e. blockIdx.y goes from 0 -> n-1. blockIdx.x goes from 0 -> p/word_size -1 
    propagate<T><<<gridDim,blockDim>>>(d_A, d_W, d_O);

    A_list[i+1] = new T[n*layers[i+1]+1](); //this auto-initialises all to zero. FOR SOME REASON, IF I DONT INITIALISE THE LAST ELEMENT (be that thru either initialising the whole array, or just the last element), THE FIRST TIME DOING THE FORWARD PASS FAILS TO TRANSFER THE LAST ELEMENT.
    // A_list[i+1][n*layers[i+1]] = 0;  //for really large layers, it might be faster just to init the last element. this bug is super annoying.
    if (hipSuccess != hipMemcpy(A_list[i+1], d_O, n*A_bytesizes[i+1]+1, hipMemcpyDeviceToHost) ) {throw std::runtime_error("CUDA memcpy failed");};

    return;
}